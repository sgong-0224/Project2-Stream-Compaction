#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        __global__ void kernNaiveScan(int n, int offset, int* odata, const int* idata)
        {
            int idx = threadIdx.x + blockIdx.x * blockDim.x;
            if (idx >= n)
                return;
            odata[idx] = idata[idx];
            if (idx >= offset)
                odata[idx] += idata[idx - offset];
        }
        __global__ void kernShift(int n, int* odata, const int* idata)
        {
            int idx = threadIdx.x + blockIdx.x * blockDim.x;
            if (idx >= n)
                return;
            odata[idx] = idx == 0 ? 0 : idata[idx - 1];
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_idata;
            int* dev_odata;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, 0);

            int minBlockSize = prop.warpSize, maxBlockSize = prop.maxThreadsPerBlock, SMCount = prop.multiProcessorCount;
            int blockSize = std::max(minBlockSize, std::min(n, maxBlockSize));
            int gridSize = (int)ceil((float)(n + blockSize - 1) / (float)blockSize);

            int max_d = ilog2ceil(n);
            timer().startGpuTimer();
            // TODO
            for (int d = 0; d < max_d; ++d) {
                kernNaiveScan <<<gridSize, blockSize>>>(n, 1<<d, dev_odata, dev_idata);
                std::swap(dev_idata, dev_odata);
            }
            kernShift <<<gridSize, blockSize >>> (n, dev_odata, dev_idata);
            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_idata);
            hipFree(dev_odata);
        }
    }
}
